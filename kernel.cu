#include "hip/hip_runtime.h"
// josecarlos.garciaortega@alum.uca.es
#include "hip/hip_runtime.h"
#include ""

#include <stdio.h>
#include <stdlib.h>
#include <string.h>

typedef enum {
	invalid_plaintext,
	invalid_key,
	cuda_fail,
	successful
} cipher_status;

__global__ void internal_vigenere(char * dev_text, const char * dev_key, int keyLen, int len) {
	unsigned int i;
	char c;
	i = threadIdx.x + blockIdx.x * blockDim.x;
	
	c = dev_text[i];
	if (c >= 'a' && c <= 'z')
		c += 'A' - 'a';
	else if (c < 'A' || c > 'Z')
		return;

	dev_text[i] = (c - dev_key[i % keyLen] + 26) % 26 + 'A';
}
cipher_status generate_vigenere(const char * plainText, char * key, char * cipherText) {
	int blockSize, minGridSize, gridSize, i, keyLen, len;
	char * dev_text, *dev_key;
	hipError_t cudaStatus;

	keyLen = strlen(key);
	len = strlen(plainText);

	for (i = 0; i < keyLen; ++i)
	{
		if (key[i] >= 'A' && key[i] <= 'Z')
			key[i] = key[i];
		else if (key[i] >= 'a' && key[i] <= 'z')
			key[i] = key[i] + 'A' - 'a';
	}
	
	// Start cuda interface
	cudaStatus = hipSetDevice(0);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipSetDevice failed!  Do you have a CUDA-capable GPU installed?");
		return cuda_fail;
	}

	cudaStatus = hipMalloc((void**)&dev_text, (len+1)*sizeof(char));
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMalloc failed!");
		return cuda_fail;
	}

	cudaStatus = hipMalloc((void**)&dev_key, (keyLen + 1)*sizeof(char));
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMalloc failed!");
		return cuda_fail;
	}

	cudaStatus = hipMemcpy(dev_text, plainText, (len+1)* sizeof(char), hipMemcpyHostToDevice);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMemcpy failed!");
		return cuda_fail;
	}

	cudaStatus = hipMemcpy(dev_key, key, (keyLen + 1)* sizeof(char), hipMemcpyHostToDevice);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMemcpy failed!");
		return cuda_fail;
	}

	hipOccupancyMaxPotentialBlockSize(&minGridSize, &blockSize, internal_vigenere, 0, len);
	gridSize = (len + blockSize - 1) / blockSize;

	internal_vigenere << <gridSize, blockSize >> >(dev_text, dev_key, keyLen, len);



	// Check for any errors launching the kernel
	cudaStatus = hipGetLastError();
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "vigenere launch failed: %s\n", hipGetErrorString(cudaStatus));
		return cuda_fail;
	}

	// hipDeviceSynchronize waits for the kernel to finish, and returns
	// any errors encountered during the launch.
	cudaStatus = hipDeviceSynchronize();
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipDeviceSynchronize returned error code %d after launching jacobi!\n", cudaStatus);
		return cuda_fail;
	}

	cudaStatus = hipMemcpy(cipherText, dev_text, (len+1)* sizeof(char), hipMemcpyDeviceToHost);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMemcpy failed!");
		return cuda_fail;
	}

	return successful;
}


cipher_status cipher_vigenere (const char * plainText,  char * key, char * cipherText) {
	return generate_vigenere(plainText, key, cipherText);
}

cipher_status decipher_vigenere(const char * plainText, char * key, char * cipherText) {
	char * newKey;
	int keyLen, i;

	keyLen = strlen(key);

	newKey = (char*)malloc(sizeof(char)*(keyLen + 1));

	for (i = 0; i < keyLen; i++) newKey[i] = -key[i];
	newKey[i] = 0;

	return generate_vigenere(plainText, newKey, cipherText);
}

void cipherfile(char * key, const char * src, const char * dst) {
	FILE *f;
	long fsize;
	char * plainText, *cipherText;
	// http://stackoverflow.com/questions/14002954/c-programming-how-to-read-the-whole-file-contents-into-a-buffer
	f = fopen(src, "rb");
	fseek(f, 0, SEEK_END);
	fsize = ftell(f);
	fseek(f, 0, SEEK_SET); 

    plainText = (char*)malloc(fsize + 1);
	cipherText = (char*)malloc(fsize + 1);
	fread(plainText, fsize, 1, f);
	fclose(f);
	plainText[fsize] = 0;

	cipher_vigenere(plainText, key, cipherText);

	f = fopen(dst, "w");
	fwrite(cipherText, sizeof(char), fsize + 1, f);
	fclose(f);
}


void decipherfile(char * key, const char * src, const char * dst) {
	FILE *f;
	long fsize;
	char * plainText, *cipherText;
	// http://stackoverflow.com/questions/14002954/c-programming-how-to-read-the-whole-file-contents-into-a-buffer
	f = fopen(src, "rb");
	fseek(f, 0, SEEK_END);
	fsize = ftell(f);
	fseek(f, 0, SEEK_SET);

	plainText = (char*)malloc(fsize + 1);
	cipherText = (char*)malloc(fsize + 1);
	fread(plainText, fsize, 1, f);
	fclose(f);
	plainText[fsize] = 0;

	decipher_vigenere(plainText, key, cipherText);

	f = fopen(dst, "w");
	fwrite(cipherText, sizeof(char), fsize + 1, f);
	fclose(f);
}

int main(int argc, char ** args)
{
	puts("Jose Carlos Garcia - josecarlos.garciaortega@alum.uca.es - 2016");

	if (!strcmp(args[1], "cf") && argc == 5)
		cipherfile(args[2], args[3], args[4]);
	if (!strcmp(args[1], "df") && argc == 5)
		decipherfile(args[2], args[3], args[4]);


	return 0;
}